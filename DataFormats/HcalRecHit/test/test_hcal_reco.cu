#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "DataFormats/HcalRecHit/interface/HBHERecHit.h"

__global__ void kernel_test_hcal_rechits(HBHERecHit* other) {
    HBHERecHit rh(HcalDetId(0), 10.0f, 10.0f, 10.0f);
    other->setEnergy(rh.energy());
    other->setTime(rh.time());
    other->setTimeFalling(rh.timeFalling());
}

void test_hcal_rechits() {
    auto check_error = [](auto code) {
        if (code != hipSuccess)
            std::cout << hipGetErrorString(code) << std::endl;
    };

    HBHERecHit h_rh, h_rh_test{HcalDetId(0), 10.0f, 10.0f, 10.0f};
    HBHERecHit *d_rh;

    hipMalloc((void**)&d_rh, sizeof(HBHERecHit));
    hipMemcpy(d_rh, &h_rh, sizeof(HBHERecHit), hipMemcpyHostToDevice);
    kernel_test_hcal_rechits<<<1,1>>>(d_rh);
    hipDeviceSynchronize();
    check_error(hipGetLastError());
    hipMemcpy(&h_rh, d_rh, sizeof(HBHERecHit), hipMemcpyDeviceToHost);

    std::cout << h_rh << std::endl;
    std::cout << h_rh_test << std::endl;
    assert(h_rh.energy() == h_rh_test.energy());
    assert(h_rh.time() == h_rh_test.time());
    assert(h_rh.timeFalling() == h_rh_test.timeFalling());
    assert(h_rh.chi2() == h_rh_test.chi2());
}

int main(int argc, char ** argv) {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    std::cout << "nDevices = " << nDevices << std::endl;

    if (nDevices > 0) {
        test_hcal_rechits();

        std::cout << "all good" << std::endl;
    }

    return 0;
}

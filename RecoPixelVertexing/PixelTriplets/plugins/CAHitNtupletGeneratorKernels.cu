#include "hip/hip_runtime.h"
#include "RecoPixelVertexing/PixelTriplets/plugins/CAHitNtupletGeneratorKernelsImpl.h"
#include <mutex>

// #define NTUPLE_DEBUG
// #define GPU_DEBUG

template <typename TrackerTraits>
void CAHitNtupletGeneratorKernelsGPU<TrackerTraits>::launchKernels(HitsOnCPU const &hh,
                                                                   TkSoA *tracks_d,
                                                                   hipStream_t cudaStream) {
  using namespace gpuPixelDoublets;
  using namespace caHitNtupletGeneratorKernels;
  // these are pointer on GPU!
  auto *tuples_d = &tracks_d->hitIndices;
  auto *detId_d = &tracks_d->detIndices;
  auto *quality_d = tracks_d->qualityData();

  // zero tuples
  cms::cuda::launchZero(tuples_d, cudaStream);

  int32_t nhits = hh.nHits();

#ifdef NTUPLE_DEBUG
  std::cout << "start tuple building. N hits " << nhits << std::endl;
  if (nhits < 2)
    std::cout << "too few hits " << nhits << std::endl;
#endif

  //
  // applying conbinatoric cleaning such as fishbone at this stage is too expensive
  //

  auto nthTot = 64;
  auto stride = 4;
  auto blockSize = nthTot / stride;
  auto numberOfBlocks = this->nDoubletBlocks(blockSize);
  auto rescale = numberOfBlocks / 65536;
  blockSize *= (rescale + 1);
  numberOfBlocks = this->nDoubletBlocks(blockSize);
  assert(numberOfBlocks < 65536);
  assert(blockSize > 0 && 0 == blockSize % 16);
  dim3 blks(1, numberOfBlocks, 1);
  dim3 thrs(stride, blockSize, 1);

  kernel_connect<TrackerTraits>
      <<<blks, thrs, 0, cudaStream>>>(this->device_hitTuple_apc_,
                                      this->device_hitToTuple_apc_,  // needed only to be reset, ready for next kernel
                                      hh.view(),
                                      this->device_theCells_.get(),
                                      this->device_nCells_,
                                      this->device_theCellNeighbors_.get(),
                                      this->isOuterHitOfCell_,
                                      this->params_.caParams_);

  cudaCheck(hipGetLastError());

  // do not run the fishbone if there are hits only in BPIX1
  if (nhits > this->isOuterHitOfCell_.offset && this->params_.earlyFishbone_) {
    auto nthTot = 128;
    auto stride = 16;
    auto blockSize = nthTot / stride;
    auto numberOfBlocks = (nhits - this->isOuterHitOfCell_.offset + blockSize - 1) / blockSize;
    dim3 blks(1, numberOfBlocks, 1);
    dim3 thrs(stride, blockSize, 1);
    fishbone<TrackerTraits><<<blks, thrs, 0, cudaStream>>>(
        hh.view(), this->device_theCells_.get(), this->device_nCells_, this->isOuterHitOfCell_, nhits, false);
    cudaCheck(hipGetLastError());
  }

  blockSize = 64;
  numberOfBlocks = (3 * this->params_.cellCuts_.maxNumberOfDoublets_ / 4 + blockSize - 1) / blockSize;
  kernel_find_ntuplets<TrackerTraits><<<numberOfBlocks, blockSize, 0, cudaStream>>>(hh.view(),
                                                                                    this->device_theCells_.get(),
                                                                                    this->device_nCells_,
                                                                                    this->device_theCellTracks_.get(),
                                                                                    tuples_d,
                                                                                    this->device_hitTuple_apc_,
                                                                                    quality_d,
                                                                                    this->params_.caParams_);
#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif
  if (this->params_.doStats_)
    kernel_mark_used<TrackerTraits>
        <<<numberOfBlocks, blockSize, 0, cudaStream>>>(this->device_theCells_.get(), this->device_nCells_);
  cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  blockSize = 128;
  numberOfBlocks = (HitContainer::ctNOnes() + blockSize - 1) / blockSize;

  cms::cuda::finalizeBulk<<<numberOfBlocks, blockSize, 0, cudaStream>>>(this->device_hitTuple_apc_, tuples_d);

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  kernel_fillHitDetIndices<TrackerTraits><<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, hh.view(), detId_d);
  cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif
  kernel_fillNLayers<TrackerTraits><<<numberOfBlocks, blockSize, 0, cudaStream>>>(tracks_d, this->device_hitTuple_apc_);
  cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  // remove duplicates (tracks that share a doublet)
  numberOfBlocks = this->nDoubletBlocks(blockSize);

  kernel_earlyDuplicateRemover<TrackerTraits><<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      this->device_theCells_.get(), this->device_nCells_, tracks_d, quality_d, this->params_.dupPassThrough_);
  cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  blockSize = 128;
  numberOfBlocks = (3 * TrackerTraits::maxNumberOfTuples / 4 + blockSize - 1) / blockSize;
  kernel_countMultiplicity<TrackerTraits>
      <<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, quality_d, this->device_tupleMultiplicity_.get());
  cms::cuda::launchFinalize(this->device_tupleMultiplicity_.get(), cudaStream);
  kernel_fillMultiplicity<TrackerTraits>
      <<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, quality_d, this->device_tupleMultiplicity_.get());
  cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  // do not run the fishbone if there are hits only in BPIX1
  if (nhits > this->isOuterHitOfCell_.offset && this->params_.lateFishbone_) {
    auto nthTot = 128;
    auto stride = 16;
    auto blockSize = nthTot / stride;
    auto numberOfBlocks = (nhits - this->isOuterHitOfCell_.offset + blockSize - 1) / blockSize;
    dim3 blks(1, numberOfBlocks, 1);
    dim3 thrs(stride, blockSize, 1);
    fishbone<TrackerTraits><<<blks, thrs, 0, cudaStream>>>(
        hh.view(), this->device_theCells_.get(), this->device_nCells_, this->isOuterHitOfCell_, nhits, true);
    cudaCheck(hipGetLastError());
  }

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  // free space asap
  // this->device_isOuterHitOfCell_.reset();
}

template <typename TrackerTraits>
void CAHitNtupletGeneratorKernelsGPU<TrackerTraits>::buildDoublets(HitsOnCPU const &hh, hipStream_t stream) {
  int32_t nhits = hh.nHits();

  using namespace gpuPixelDoublets;

  using GPUCACell = GPUCACellT<TrackerTraits>;
  using OuterHitOfCell = typename GPUCACell::OuterHitOfCell;
  using CellNeighbors = typename GPUCACell::CellNeighbors;
  using CellTracks = typename GPUCACell::CellTracks;
  using OuterHitOfCellContainer = typename GPUCACell::OuterHitOfCellContainer;

  this->isOuterHitOfCell_ = OuterHitOfCell{this->device_isOuterHitOfCell_.get(), hh.offsetBPIX2()};

#ifdef NTUPLE_DEBUG
  std::cout << "building Doublets out of " << nhits << " Hits" << std::endl;
#endif

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  // in principle we can use "nhits" to heuristically dimension the workspace...
  this->device_isOuterHitOfCell_ =
      cms::cuda::make_device_unique<OuterHitOfCellContainer[]>(std::max(1, nhits - hh.offsetBPIX2()), stream);
  assert(this->device_isOuterHitOfCell_.get());

  this->isOuterHitOfCell_ = OuterHitOfCell{this->device_isOuterHitOfCell_.get(), hh.offsetBPIX2()};

  this->cellStorage_ =
      cms::cuda::make_device_unique<unsigned char[]>(TrackerTraits::maxNumOfActiveDoublets * sizeof(CellNeighbors) +
                                                         TrackerTraits::maxNumOfActiveDoublets * sizeof(CellTracks),
                                                     stream);
  this->device_theCellNeighborsContainer_ = (CellNeighbors *)this->cellStorage_.get();
  this->device_theCellTracksContainer_ =
      (CellTracks *)(this->cellStorage_.get() + TrackerTraits::maxNumOfActiveDoublets * sizeof(CellNeighbors));

  {
    int threadsPerBlock = 128;
    // at least one block!
    int blocks = (std::max(1, nhits - hh.offsetBPIX2()) + threadsPerBlock - 1) / threadsPerBlock;
    initDoublets<TrackerTraits><<<blocks, threadsPerBlock, 0, stream>>>(this->isOuterHitOfCell_,
                                                                        nhits,
                                                                        this->device_theCellNeighbors_.get(),
                                                                        this->device_theCellNeighborsContainer_,
                                                                        this->device_theCellTracks_.get(),
                                                                        this->device_theCellTracksContainer_);
    cudaCheck(hipGetLastError());
  }

  this->device_theCells_ =
      cms::cuda::make_device_unique<GPUCACell[]>(this->params_.cellCuts_.maxNumberOfDoublets_, stream);

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

  if (0 == nhits)
    return;  // protect against empty events

  // take all layer pairs into account
  auto nActualPairs = this->params_.nPairs();

  int stride = 4;
  int threadsPerBlock = TrackerTraits::getDoubletsFromHistoMaxBlockSize / stride;
  int blocks = (4 * nhits + threadsPerBlock - 1) / threadsPerBlock;
  dim3 blks(1, blocks, 1);
  dim3 thrs(stride, threadsPerBlock, 1);
  getDoubletsFromHisto<TrackerTraits><<<blks, thrs, 0, stream>>>(this->device_theCells_.get(),
                                                                 this->device_nCells_,
                                                                 this->device_theCellNeighbors_.get(),
                                                                 this->device_theCellTracks_.get(),
                                                                 hh.view(),
                                                                 this->isOuterHitOfCell_,
                                                                 nActualPairs,
                                                                 this->params_.cellCuts_);
  cudaCheck(hipGetLastError());

#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif
}

template <typename TrackerTraits>
void CAHitNtupletGeneratorKernelsGPU<TrackerTraits>::classifyTuples(HitsOnCPU const &hh,
                                                                    TkSoA *tracks_d,
                                                                    hipStream_t cudaStream) {
  using namespace caHitNtupletGeneratorKernels;

  // these are pointer on GPU!
  auto const *tuples_d = &tracks_d->hitIndices;
  auto *quality_d = tracks_d->qualityData();

  int32_t nhits = hh.nHits();

  auto blockSize = 64;

  // classify tracks based on kinematics
  auto numberOfBlocks = this->nQuadrupletBlocks(blockSize);
  kernel_classifyTracks<TrackerTraits>
      <<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, tracks_d, this->params_.qualityCuts_, quality_d);

  if (this->params_.lateFishbone_) {
    // apply fishbone cleaning to good tracks
    numberOfBlocks = this->nDoubletBlocks(blockSize);
    kernel_fishboneCleaner<TrackerTraits>
        <<<numberOfBlocks, blockSize, 0, cudaStream>>>(this->device_theCells_.get(), this->device_nCells_, quality_d);
    cudaCheck(hipGetLastError());
  }

  // mark duplicates (tracks that share a doublet)
  numberOfBlocks = this->nDoubletBlocks(blockSize);
  kernel_fastDuplicateRemover<TrackerTraits><<<numberOfBlocks, blockSize, 0, cudaStream>>>(
      this->device_theCells_.get(), this->device_nCells_, tracks_d, this->params_.dupPassThrough_);
  cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
  cudaCheck(hipDeviceSynchronize());
#endif

  if (this->params_.doSharedHitCut_ || this->params_.doStats_) {
    // fill hit->track "map"
    assert(this->hitToTupleView_.offSize > nhits);
    numberOfBlocks = this->nQuadrupletBlocks(blockSize);
    kernel_countHitInTracks<TrackerTraits>
        <<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, quality_d, this->device_hitToTuple_.get());
    cudaCheck(hipGetLastError());
    assert((this->hitToTupleView_.assoc == this->device_hitToTuple_.get()) &&
           (this->hitToTupleView_.offStorage == this->device_hitToTupleStorage_.get()) &&
           (this->hitToTupleView_.offSize > 0));
    cms::cuda::launchFinalize(this->hitToTupleView_, cudaStream);
    cudaCheck(hipGetLastError());
    kernel_fillHitInTracks<TrackerTraits>
        <<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, quality_d, this->device_hitToTuple_.get());
    cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
    cudaCheck(hipDeviceSynchronize());
#endif
  }

  if (this->params_.doSharedHitCut_) {
    // mark duplicates (tracks that share at least one hit)
    numberOfBlocks = (this->hitToTupleView_.offSize + blockSize - 1) / blockSize;

    kernel_rejectDuplicate<TrackerTraits>
        <<<numberOfBlocks, blockSize, 0, cudaStream>>>(tracks_d,
                                                       quality_d,
                                                       this->params_.minHitsForSharingCut_,
                                                       this->params_.dupPassThrough_,
                                                       this->device_hitToTuple_.get());

    kernel_sharedHitCleaner<TrackerTraits>
        <<<numberOfBlocks, blockSize, 0, cudaStream>>>(hh.view(),
                                                       tracks_d,
                                                       quality_d,
                                                       this->params_.minHitsForSharingCut_,
                                                       this->params_.dupPassThrough_,
                                                       this->device_hitToTuple_.get());

    if (this->params_.useSimpleTripletCleaner_) {
      kernel_simpleTripletCleaner<TrackerTraits>
          <<<numberOfBlocks, blockSize, 0, cudaStream>>>(tracks_d,
                                                         quality_d,
                                                         this->params_.minHitsForSharingCut_,
                                                         this->params_.dupPassThrough_,
                                                         this->device_hitToTuple_.get());
    } else {
      kernel_tripletCleaner<TrackerTraits>
          <<<numberOfBlocks, blockSize, 0, cudaStream>>>(tracks_d,
                                                         quality_d,
                                                         this->params_.minHitsForSharingCut_,
                                                         this->params_.dupPassThrough_,
                                                         this->device_hitToTuple_.get());
    }
    cudaCheck(hipGetLastError());
#ifdef GPU_DEBUG
    cudaCheck(hipDeviceSynchronize());
#endif
  }

  if (this->params_.doStats_) {
    numberOfBlocks = (std::max(nhits, int(this->params_.cellCuts_.maxNumberOfDoublets_)) + blockSize - 1) / blockSize;
    kernel_checkOverflows<TrackerTraits>
        <<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d,
                                                       this->device_tupleMultiplicity_.get(),
                                                       this->device_hitToTuple_.get(),
                                                       this->device_hitTuple_apc_,
                                                       this->device_theCells_.get(),
                                                       this->device_nCells_,
                                                       this->device_theCellNeighbors_.get(),
                                                       this->device_theCellTracks_.get(),
                                                       this->isOuterHitOfCell_,
                                                       nhits,
                                                       this->params_.cellCuts_.maxNumberOfDoublets_,
                                                       this->counters_);
    cudaCheck(hipGetLastError());
  }

  if (this->params_.doStats_) {
    // counters (add flag???)
    numberOfBlocks = (this->hitToTupleView_.offSize + blockSize - 1) / blockSize;
    kernel_doStatsForHitInTracks<TrackerTraits>
        <<<numberOfBlocks, blockSize, 0, cudaStream>>>(this->device_hitToTuple_.get(), this->counters_);
    cudaCheck(hipGetLastError());
    numberOfBlocks = (3 * TrackerTraits::maxNumberOfQuadruplets / 4 + blockSize - 1) / blockSize;
    kernel_doStatsForTracks<TrackerTraits>
        <<<numberOfBlocks, blockSize, 0, cudaStream>>>(tuples_d, quality_d, this->counters_);
    cudaCheck(hipGetLastError());
  }
#ifdef GPU_DEBUG
  hipDeviceSynchronize();
  cudaCheck(hipGetLastError());
#endif

#ifdef DUMP_GPU_TK_TUPLES
  static std::atomic<int> iev(0);
  static std::mutex lock;
  {
    std::lock_guard<std::mutex> guard(lock);
    ++iev;
    for (int k = 0; k < 20000; k += 500) {
      kernel_print_found_ntuplets<TrackerTraits><<<1, 32, 0, cudaStream>>>(
          hh.view(), tuples_d, tracks_d, quality_d, this->device_hitToTuple_.get(), k, k + 500, iev);
      hipDeviceSynchronize();
    }
    kernel_print_found_ntuplets<TrackerTraits><<<1, 32, 0, cudaStream>>>(
        hh.view(), tuples_d, tracks_d, quality_d, this->device_hitToTuple_.get(), 20000, 1000000, iev);
    hipDeviceSynchronize();
    // hipStreamSynchronize(cudaStream);
  }
#endif
}

template <typename TrackerTraits>
void CAHitNtupletGeneratorKernelsGPU<TrackerTraits>::printCounters(Counters const *counters) {
  caHitNtupletGeneratorKernels::kernel_printCounters<<<1, 1>>>(counters);
}

template class CAHitNtupletGeneratorKernelsGPU<pixelTopology::Phase1>;
template class CAHitNtupletGeneratorKernelsGPU<pixelTopology::Phase2>;
